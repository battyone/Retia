﻿#include "CApi.h"

RMSPropOptimizer* CreateRMSPropOptimizer(float learningRate, float momentum, float decayRate, float weightDecay)
{
	return new RMSPropOptimizer(learningRate, momentum, decayRate, weightDecay);
}

void DestroyOptimizer(OptimizerBase* optimizer)
{
	delete optimizer;
}

void SetLearningRate(OptimizerBase* optimizer, float learningRate)
{
	optimizer->setLearningRate(learningRate);
}

LayeredNet* CreateLayeredNetwork(int inputSize, int outputSize, int batchSize, int seqLen)
{
	return new LayeredNet(inputSize, outputSize, batchSize, seqLen);
}

void DestroyLayeredNetwork(LayeredNet* network)
{
	delete network;
}

void SetNetworkOptimizer(LayeredNet* network, OptimizerBase* optimizer)
{
	network->setOptimizer(optimizer);
}

void AddNetworkLayer(LayeredNet* network, LayerBase* layer)
{
	network->AddLayer(layer);
}

LinearLayer* CreateLinearLayer(int inputSize, int outSize, int batchSize, int seqLen)
{
	return new LinearLayer(inputSize, outSize, batchSize, seqLen);
}

GruLayer* CreateGruLayer(int inputSize, int hSize, int layers, int batchSize, int seqLen)
{
	return new GruLayer(inputSize, hSize, layers, batchSize, seqLen);
}

SoftmaxLayer* CreateSoftmaxLayer(int inSize, int batchSize, int seqLen)
{
	return new SoftmaxLayer(inSize, batchSize, seqLen);
}

void TransferLayerStatesToDevice(LayerBase* layer, WeightDefinition *weights, int count)
{
	auto states = GetWeightSyncContainers(weights, count);

	layer->TransferStatesToDevice(states);

	DestroyWeightSyncContainers(states);
}

void TransferLayerStatesToHost(LayerBase* layer, WeightDefinition *weights, int count)
{
	auto states = GetWeightSyncContainers(weights, count);

	layer->TransferStatesToHost(states);

	DestroyWeightSyncContainers(states);
}

double TrainSequence(LayeredNet* net, MatrixDefinition* inputs, MatrixDefinition* targets, int count)
{
	auto in = GetMatrixPointers(inputs, count);
	auto targ = GetMatrixPointers(targets, count);

	double result = net->TrainSequence(in, targ);

	DestroyMatrixPointers(in);
	DestroyMatrixPointers(targ);

	return result;
}

std::vector<WeightSyncContainer*> GetWeightSyncContainers(WeightDefinition* weights, int count)
{
	std::vector<WeightSyncContainer*> result;

	for (int i = 0; i < count; ++i)
	{
		auto cur = weights[i];
		result.push_back(new WeightSyncContainer(cur.Rows, cur.Columns, cur.SeqLength, cur.WeightPtr, cur.GradPtr, cur.Cache1Ptr, cur.Cache2Ptr, cur.CacheMPtr));
	}

	return result;
}

void DestroyWeightSyncContainers(std::vector<WeightSyncContainer*>& containers)
{
	for (int i = 0; i < containers.size(); ++i)
	{
		delete containers[i];
	}
}

std::vector<HostMatrixPtr*> GetMatrixPointers(MatrixDefinition* matrices, int matrixCount)
{
	std::vector<HostMatrixPtr*> result;

	for (int i = 0; i < matrixCount; ++i)
	{
		auto cur = matrices[i];
		result.push_back(new HostMatrixPtr(cur.Rows, cur.Columns, cur.SeqLength, cur.Pointer));
	}

	return result;
}

void DestroyMatrixPointers(std::vector<HostMatrixPtr*>& ptrs)
{
	for (int i = 0; i < ptrs.size(); ++i)
	{
		delete ptrs[i];
	}
}


