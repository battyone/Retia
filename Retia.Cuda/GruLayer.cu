﻿#include "GruLayer.h"
#include "CudaContext.h"
#include "Algorithms.h"
#include "Helpers.h"
#include <iostream>
#include <thrust/device_ptr.h>
#include <thrust/copy.h>

using std::make_unique;
using std::cout;
using std::endl;

GruLayer::GruLayer(int inSize, int hSize, int layers, int batchSize, int seqLength)
	: LayerBase(inSize, hSize, batchSize, seqLength), 
	_hSize(hSize),
	_layers(layers)
{
	InitLayers();
}

/*
* States indexes for each layer:
* 0  - Wxr
* 1  - Wxz
* 2  - Wxh
*
* 3  - Whr
* 4  - Whz
* 5  - Whh
*
* 6  - bxr
* 7  - bxz
* 8  - bxh
*
* 9  - bhr
* 10 - bhz
* 11 - bhh
*/
void GruLayer::TransferStatesToDevice(std::vector<WeightSyncContainer*>& states)
{
	if (states.size() != _weights.size()) throw RetiaException("There should be exactly 12 state vectors for each layer");

	for (int i = 0; i < states.size(); ++i)
	{
		_weights[i]->TransferStateToDeviceLoose(*states[i]);
	}
}

void GruLayer::TransferStatesToHost(std::vector<WeightSyncContainer*>& states)
{
	if (states.size() != _weights.size()) throw RetiaException("There should be exactly 12 state vectors for each layer");

	for (int i = 0; i < states.size(); ++i)
	{
		_weights[i]->TransferStateToHostLoose(*states[i]);
	}
}

void GruLayer::ForwardSequence(DeviceMatrix& input)
{
	/*cout << "GRU input" << endl;
	PrintMatrix(input);*/

	// Forward through RNN
	auto result = hipdnnRNNForwardTraining(CudaContext::cudnnHandle(), _rnnDesc, _seqLen,
		*_xTensor, input.raw_ptr(),
		*_hxTensor, _hxTensor->device_ptr(),
		*_cxTensor, nullptr,
		*_wFilter, _w->weight().raw_ptr(),
		*_yTensor, _output->raw_ptr(),
		*_hyTensor, _hyTensor->device_ptr(),
		*_cyTensor, nullptr,
		_workspace->device_ptr(), _workspace->size(),
		_reserve->device_ptr(), _reserve->size());
	if (result != HIPDNN_STATUS_SUCCESS)
	{
		throw CuDnnException(result);
	}

	/*cout << "GRU output" << endl;
	PrintMatrix(*_output);*/
}

void GruLayer::BackpropSequence(DeviceMatrix& input, DeviceMatrix& outSens)
{
	// RNN backprop
	auto result = hipdnnRNNBackwardData(CudaContext::cudnnHandle(), _rnnDesc, _seqLen,
		*_yTensor, _output->raw_ptr(),
		*_dyTensor, outSens.raw_ptr(),
		*_dhyTensor, _dhyTensor->device_ptr(),
		*_dcyTensor, nullptr,
		*_wFilter, _w->weight().raw_ptr(),
		*_hxTensor, _hxTensor->device_ptr(),
		*_cxTensor, nullptr,
		*_dxTensor, _sensitivity->raw_ptr(),
		*_dhxTensor, _dhxTensor->device_ptr(),
		*_dcxTensor, nullptr,
		_workspace->device_ptr(), _workspace->size(),
		_reserve->device_ptr(), _reserve->size());
	if (result != HIPDNN_STATUS_SUCCESS)
	{
		throw CuDnnException(result);
	}

	_w->ClearGradient();
	result = hipdnnRNNBackwardWeights(CudaContext::cudnnHandle(), _rnnDesc, _seqLen,
		*_xTensor, input.raw_ptr(),
		*_hxTensor, _hxTensor->device_ptr(),
		*_yTensor, _output->raw_ptr(),
		_workspace->device_ptr(), _workspace->size(),
		*_dwFilter, _w->gradient().raw_ptr(),
		_reserve->device_ptr(), _reserve->size());
	if (result != HIPDNN_STATUS_SUCCESS)
	{
		throw CuDnnException(result);
	}

	Algorithms::Clamp(_w->gradient(), 5.0f);
}

void GruLayer::Optimize(OptimizerBase& optimizer)
{
	optimizer.Optimize(*_w);
}

void GruLayer::ResetMemory()
{
	_hxTensor->ZeroMemory();
	_hyTensor->ZeroMemory();
	_dhxTensor->ZeroMemory();
	_dhyTensor->ZeroMemory();
}

void GruLayer::ResetOptimizerCache()
{
	_w->ClearCache();
}

void GruLayer::InitLayers()
{
	// Allocate RNN in/out tensors and GPU memory
	_xTensor = make_unique<CuDnnNdTensorArray>(_batchSize, _inputSize, 1, _seqLen, false);
	_dxTensor = make_unique<CuDnnNdTensorArray>(_batchSize, _inputSize, 1, _seqLen, false);
	_yTensor = make_unique<CuDnnNdTensorArray>(_batchSize, _hSize, 1, _seqLen, false);
	_dyTensor = make_unique<CuDnnNdTensorArray>(_batchSize, _hSize, 1, _seqLen, false);

	_output = make_unique<DeviceMatrix>(_hSize, _batchSize, _seqLen);
	_sensitivity = make_unique<DeviceMatrix>(_inputSize, _batchSize, _seqLen);
	_output->ZeroMemory();
	_sensitivity->ZeroMemory();

	// Allocate RNN hidden state tensors and GPU memory
	_hxTensor = make_unique<CuDnnNdTensor>(_layers, _batchSize, _hSize);
	_hyTensor = make_unique<CuDnnNdTensor>(_layers, _batchSize, _hSize);
	_dhxTensor = make_unique<CuDnnNdTensor>(_layers, _batchSize, _hSize);
	_dhyTensor = make_unique<CuDnnNdTensor>(_layers, _batchSize, _hSize);
	_cxTensor = make_unique<CuDnnNdTensor>(_layers, _batchSize, _hSize, false);
	_cyTensor = make_unique<CuDnnNdTensor>(_layers, _batchSize, _hSize, false);
	_dcxTensor = make_unique<CuDnnNdTensor>(_layers, _batchSize, _hSize, false);
	_dcyTensor = make_unique<CuDnnNdTensor>(_layers, _batchSize, _hSize, false);

	// Allocate dropout and RNN descriptors
	_dropoutDesc.Create();

	size_t dropoutSz;
	auto result = hipdnnDropoutGetStatesSize(CudaContext::cudnnHandle(), &dropoutSz);
	if (result != HIPDNN_STATUS_SUCCESS)
	{
		throw CuDnnException(result);
	}

	_dropoutStates = make_unique<CudaMemoryBlock>(dropoutSz);
	result = hipdnnSetDropoutDescriptor(_dropoutDesc, CudaContext::cudnnHandle(), 0.0f, _dropoutStates->device_ptr(), dropoutSz, 1337ull);
	if (result != HIPDNN_STATUS_SUCCESS)
	{
		throw CuDnnException(result);
	}

	// Create RNN descriptor
	_rnnDesc.Create();
	result = hipdnnSetRNNDescriptor(_rnnDesc, _hSize, _layers, _dropoutDesc, HIPDNN_LINEAR_INPUT, HIPDNN_UNIDIRECTIONAL, HIPDNN_GRU, HIPDNN_DATA_FLOAT);
	if (result != HIPDNN_STATUS_SUCCESS)
	{
		throw CuDnnException(result);
	}

	// Allocate RNN parameters
	size_t weightsSize;
	result = hipdnnGetRNNParamsSize(CudaContext::cudnnHandle(), _rnnDesc, (*_xTensor)[0], &weightsSize, HIPDNN_DATA_FLOAT);
	if (result != HIPDNN_STATUS_SUCCESS)
	{
		throw CuDnnException(result);
	}

	int wLen = (int)(weightsSize / sizeof(float));
	_wFilter = make_unique<CuDnnFilter>(wLen, 1, 1, false);
	_dwFilter = make_unique<CuDnnFilter>(wLen, 1, 1, false);
	_w = make_unique<NeuroWeight>(wLen, 1, 1);
	InitWeights();

	// Allocate workspace and reserve
	size_t wsSize, reserveSize;
	result = hipdnnGetRNNWorkspaceSize(CudaContext::cudnnHandle(), _rnnDesc, _seqLen, *_xTensor, &wsSize);
	if (result != HIPDNN_STATUS_SUCCESS)
	{
		throw CuDnnException(result);
	}

	result = hipdnnGetRNNTrainingReserveSize(CudaContext::cudnnHandle(), _rnnDesc, _seqLen, *_xTensor, &reserveSize);
	if (result != HIPDNN_STATUS_SUCCESS)
	{
		throw CuDnnException(result);
	}

	_workspace = make_unique<CudaMemoryBlock>(wsSize);
	_reserve = make_unique<CudaMemoryBlock>(reserveSize);
}

void GruLayer::InitWeights()
{
	for (int i = 0; i < _layers * 12; ++i)
	{
		_weights.push_back(std::unique_ptr<NeuroWeightPtr>());
	}

	hipdnnStatus_t result;
	for (int layer = 0; layer < _layers; layer++)
	{
		for (int linLayerID = 0; linLayerID < 6; linLayerID++) { // 6 matrices for GRU
			int matIdx = layer * 12 + linLayerID;
			int bIdx = matIdx + 6;

			hipdnnFilterDescriptor_t filterDesc;
			result = hipdnnCreateFilterDescriptor(&filterDesc);
			if (result != HIPDNN_STATUS_SUCCESS)
			{
				throw CuDnnException(result);
			}

			float *filterMemPtr;

			// Weight matrix
			result = hipdnnGetRNNLinLayerMatrixParams(CudaContext::cudnnHandle(), _rnnDesc, layer, (*_xTensor)[0], *_wFilter, _w->weight().raw_ptr(), linLayerID, filterDesc, (void**)&filterMemPtr);
			if (result != HIPDNN_STATUS_SUCCESS)
			{
				throw CuDnnException(result);
			}

			_weights[matIdx].reset(GetWeightPtr(filterDesc, filterMemPtr));

			result = hipdnnDestroyFilterDescriptor(filterDesc);
			if (result != HIPDNN_STATUS_SUCCESS)
			{
				throw CuDnnException(result);
			}

			// Bias vector
			result = hipdnnCreateFilterDescriptor(&filterDesc);
			if (result != HIPDNN_STATUS_SUCCESS)
			{
				throw CuDnnException(result);
			}

			result = hipdnnGetRNNLinLayerBiasParams(CudaContext::cudnnHandle(), _rnnDesc, layer, (*_xTensor)[0], *_wFilter, _w->weight().raw_ptr(), linLayerID, filterDesc, (void**)&filterMemPtr);
			if (result != HIPDNN_STATUS_SUCCESS)
			{
				throw CuDnnException(result);
			}

			_weights[bIdx].reset(GetWeightPtr(filterDesc, filterMemPtr));

			result = hipdnnDestroyFilterDescriptor(filterDesc);
			if (result != HIPDNN_STATUS_SUCCESS)
			{
				throw CuDnnException(result);
			}
		}
	}
}

std::tuple<int, int, int> GruLayer::GetTensorDims(hipdnnFilterDescriptor_t desc)
{
	hipdnnDataType_t dataType;
	hipdnnTensorFormat_t format;
	int nbDims;
	int dims[3];

	auto result = hipdnnGetFilterNdDescriptor(desc, 3, &dataType, &format, &nbDims, dims);
	if (result != HIPDNN_STATUS_SUCCESS)
	{
		throw CuDnnException(result);
	}

	return std::make_tuple(dims[1], dims[0], dims[2]);
}

NeuroWeightPtr* GruLayer::GetWeightPtr(hipdnnFilterDescriptor_t tensor, float* weightPtr)
{
	auto dims = GetTensorDims(tensor);
	auto delta = weightPtr - _w->weight().raw_ptr(); // We assume that gradients have the same offset as weights
	float* gradPtr = _w->gradient().raw_ptr() + delta;
	float* cache1Ptr = _w->cache1().raw_ptr() + delta;
	float* cache2Ptr = _w->cache2().raw_ptr() + delta;
	float* cacheMPtr = _w->cache_m().raw_ptr() + delta;
	return new NeuroWeightPtr(std::get<0>(dims), std::get<1>(dims), std::get<2>(dims), weightPtr, gradPtr, cache1Ptr, cache2Ptr, cacheMPtr);
}
